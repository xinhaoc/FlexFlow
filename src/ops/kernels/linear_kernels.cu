#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ffconst_utils.h"
#include "flexflow/initializer.h"
#include "flexflow/ops/kernels/decompress_kernels.h"
#include "flexflow/ops/kernels/linear_kernels.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

LinearMeta::LinearMeta(FFHandler handler,
                       int batch_size,
                       Linear const *li,
                       MemoryAllocator gpu_mem_allocator,
                       int weightSize)
    : OpMeta(handler, li), weight_ptr(nullptr) {
  DataType data_type = li->data_type;
  // allocate weight and bias in the reserve space for cpu offloading
  if (li->offload) {
    weight_ptr = gpu_mem_allocator.allocate_reserved_untyped(
        weightSize * data_type_size(data_type));
    if (li->quantization_type != DT_NONE) {
      quantized_weightSize = get_quantization_to_byte_size(
          data_type, li->quantization_type, weightSize);
      quantized_weight_ptr =
          gpu_mem_allocator.allocate_reserved<char>(quantized_weightSize);
    }
  }
  // Allocate an all-one's vector
  gpu_mem_allocator.create_legion_instance(
      reserveInst, data_type_size(data_type) * batch_size);
  one_ptr = gpu_mem_allocator.allocate_instance_untyped(
      data_type_size(data_type) * batch_size);
  int parallelism = batch_size;
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  if (data_type == DT_FLOAT) {
    Kernels::Linear::Internal::
        build_one_ptr<<<GET_BLOCKS(parallelism),
                        min(CUDA_NUM_THREADS, parallelism),
                        0,
                        stream>>>((float *)one_ptr, batch_size);
  } else if (data_type == DT_HALF) {
    Kernels::Linear::Internal::
        build_one_ptr<<<GET_BLOCKS(parallelism),
                        min(CUDA_NUM_THREADS, parallelism),
                        0,
                        stream>>>((half *)one_ptr, batch_size);
  }

  // Allocate descriptors
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
}

#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
std::mutex LinearMeta::profile_lock;
std::unordered_map<cublasAlgoConfig_t, int, cublasAlgoConfig_hasher>
    LinearMeta::algo_map;

void LinearMeta::findBestAlgoID(int m, int n, int k) {
  // return; // NO_PROF

  std::lock_guard<std::mutex> lock(profile_lock);

  hipDataType cublas_data_type = ff_to_cuda_datatype(output_type[0]);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = cublas_data_type;
#endif

  cublasAlgoConfig_t mark{1, m, n, k, (int)compute_type};
  auto iter = algo_map.find(mark);
  if (iter != algo_map.end()) {
    return;
  }

  // don't find, profile required
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  checkCUDA(hipblasSetStream(handle.blas, stream));
  checkCUDNN(hipdnnSetStream(handle.dnn, stream));
  switch (output_type[0]) {
    case DT_FLOAT:
      cudaRandomUniform(static_cast<float *>(handle.workSpace),
                        handle.workSpaceSize / data_type_size(DT_FLOAT));
      break;
    case DT_HALF:
      cudaRandomUniform(static_cast<half *>(handle.workSpace),
                        handle.workSpaceSize / data_type_size(DT_HALF));
      break;
    default:
      assert(false);
      break;
  }

  float alpha = 1.0f, beta = 0.0f;

  int startAlgo, endAlgo;
  int const ites = 100;
  struct timeval start, end;
  // TODO: only support 16F
  if (compute_type == HIPBLAS_COMPUTE_16F) {
    startAlgo = (int)CUBLAS_GEMM_DEFAULT_TENSOR_OP;
    endAlgo = (int)CUBLAS_GEMM_ALGO15_TENSOR_OP;
  } else if (compute_type == HIP_R_32F) {
    startAlgo = (int)HIPBLAS_GEMM_DEFAULT;
    endAlgo = (int)CUBLAS_GEMM_ALGO23;
  } else {
    assert(false);
  }

  printf("***Cublas Gemm Testing Begin (Linear)***\n");
  printf("\n-----------------------------\n");
  printf("GEMM test: [M: %d, K: %d, N: %d] for linear\n", m, k, n);
  // todo
  void *d_A = handle.workSpace;
  void *d_B = d_A + m * k * data_type_size(output_type[0]);
  void *d_C = d_B + k * n * data_type_size(output_type[0]);

  float exec_time = 99999.0f;
  int fast_algo = 0;

  for (int algo = startAlgo; algo <= endAlgo; algo++) {
    hipblasStatus_t status;
    hipDeviceSynchronize();
    gettimeofday(&start, NULL);
    for (int ite = 0; ite < ites; ++ite) {
      // checkCUDA(hipblasGemmEx(m->handle.blas,
      //                    HIPBLAS_OP_T,
      //                    HIPBLAS_OP_N,
      //                    out_dim,
      //                    batch_size,
      //                    in_dim,
      //                    &alpha,
      //                    m->offload ? m->weight_ptr : weight_ptr,
      //                    weight_type,
      //                    in_dim,
      //                    input_ptr,
      //                    input_type,
      //                    in_dim,
      //                    &beta,
      //                    output_ptr,
      //                    output_type,
      //                    out_dim,
      //                    compute_type,
      //                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));
      status = hipblasGemmEx(handle.blas,
                            HIPBLAS_OP_T,
                            HIPBLAS_OP_N,
                            m, // out_dim
                            n, // batch_size
                            k, // in_dim
                            &alpha,
                            d_A,
                            cublas_data_type,
                            k,
                            d_B,
                            cublas_data_type,
                            k,
                            &beta,
                            d_C,
                            cublas_data_type,
                            m,
                            compute_type,
                            static_cast<hipblasGemmAlgo_t>(algo));

      if (status != HIPBLAS_STATUS_SUCCESS) {
        break;
      }
    }
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
    if (status == HIPBLAS_STATUS_SUCCESS) {
      printf("algo_%d costs %.3fms \n", algo, diffTime(start, end) / ites);
      if (diffTime(start, end) / ites < exec_time) {
        exec_time = diffTime(start, end) / ites;
        fast_algo = algo;
      }
    }
  }

  printf("fast_algo %d costs %.3f ms\n", fast_algo, exec_time);
  algo_map.insert(std::pair<const cublasAlgoConfig_t, int>(mark, fast_algo));
}
#endif
LinearMeta::~LinearMeta(void) {
  if (reserveInst != Realm::RegionInstance::NO_INST) {
    reserveInst.destroy();
  }
}

namespace Kernels {
namespace Linear {

bool use_activation(ActiMode mode) {
  switch (mode) {
    case AC_MODE_RELU:
    case AC_MODE_SIGMOID:
    case AC_MODE_TANH:
      return true;
    case AC_MODE_NONE:
      return false;
    default:
      assert(0);
      break;
  }
  return false;
}

void init_kernel(LinearMeta *m, int batch_size, int channel) {
  if (use_activation(m->activation)) {
    hipdnnActivationMode_t mode;
    switch (m->activation) {
      case AC_MODE_RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case AC_MODE_SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
    checkCUDNN(hipdnnSetActivationDescriptor(
        m->actiDesc, mode, HIPDNN_PROPAGATE_NAN, 0.0));
    checkCUDNN(
        hipdnnSetTensor4dDescriptor(m->outputTensor,
                                   HIPDNN_TENSOR_NCHW,
                                   ff_to_cudnn_datatype(m->output_type[0]),
                                   batch_size,
                                   channel,
                                   1,
                                   1));
  }
}

void forward_kernel_wrapper(LinearMeta const *m,
                            void const *input_ptr,
                            void *output_ptr,
                            void const *weight_ptr,
                            void const *bias_ptr,
                            int in_dim,
                            int out_dim,
                            int batch_size) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));
  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  if (m->input_type[0] == DT_FLOAT) {
    Internal::forward_kernel<float>(m,
                                    input_ptr,
                                    output_ptr,
                                    weight_ptr,
                                    bias_ptr,
                                    in_dim,
                                    out_dim,
                                    batch_size,
                                    stream);
  } else if (m->input_type[0] == DT_HALF) {
    Internal::forward_kernel<half>(m,
                                   input_ptr,
                                   output_ptr,
                                   weight_ptr,
                                   bias_ptr,
                                   in_dim,
                                   out_dim,
                                   batch_size,
                                   stream);
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s [Linear] forward time = %.2lfms\n", m->op_name, elapsed);
    // print_tensor<float>((float*)input_ptr, in_dim * batch_size,
    // "[Linear:forward:input]"); print_tensor<float>((float*)weight_ptr, in_dim
    // * out_dim, "[Linear:forward:kernel]");
    // print_tensor<float>((float*)output_ptr, out_dim * batch_size,
    // "[Linear:forward:output]");
  }
}

void backward_kernel_wrapper(LinearMeta const *m,
                             void const *input_ptr,
                             void *input_grad_ptr,
                             void const *output_ptr,
                             void *output_grad_ptr,
                             void const *kernel_ptr,
                             void *kernel_grad_ptr,
                             void *bias_grad_ptr,
                             int in_dim,
                             int out_dim,
                             int batch_size) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  if (m->input_type[0] == DT_FLOAT) {
    Internal::backward_kernel<float>(m,
                                     input_ptr,
                                     input_grad_ptr,
                                     output_ptr,
                                     output_grad_ptr,
                                     kernel_ptr,
                                     kernel_grad_ptr,
                                     bias_grad_ptr,
                                     in_dim,
                                     out_dim,
                                     batch_size,
                                     stream);
  } else if (m->input_type[0] == DT_HALF) {
    Internal::backward_kernel<half>(m,
                                    input_ptr,
                                    input_grad_ptr,
                                    output_ptr,
                                    output_grad_ptr,
                                    kernel_ptr,
                                    kernel_grad_ptr,
                                    bias_grad_ptr,
                                    in_dim,
                                    out_dim,
                                    batch_size,
                                    stream);
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("%s Linear backward time = %.2lfms\n", m->op_name, elapsed);
    // print_tensor<float>(acc_output_grad.ptr, acc_output_grad.rect.volume(),
    // "[Linear:backward:output_grad]");
    // print_tensor<float>(acc_kernel_grad.ptr, acc_kernel_grad.rect.volume(),
    // "[Linear:backward:kernel_grad]"); print_tensor<1,
    // float>(acc_bias_grad.ptr, acc_bias_grad.rect,
    // "[Linear:backward:bias_grad]"); print_tensor<float>(input_grad,
    // acc_input.rect.volume(), "[Linear:backward:input_grad]");
  }
}

/*
__host__
Parameter* Linear::get_parameter(int index)
{
  if (index == 0) {
    return &weights[0];
  } else if (index == 1){
    return &weights[1];
  } else {
    assert(0);
    return NULL;
  }
}
*/
namespace Internal {

template <typename DT>
void forward_kernel(LinearMeta const *m,
                    void const *input_ptr,
                    void *output_ptr,
                    void const *weight_ptr,
                    void const *bias_ptr,
                    int in_dim,
                    int out_dim,
                    int batch_size,
                    ffStream_t stream) {
  // additional processing for uploading weights
  if (m->offload) {
    // Note that we update weight_ptr when uploading weight
    if (m->quantization_type != DT_NONE) {
      hipMemcpyAsync(m->quantized_weight_ptr,
                      weight_ptr,
                      m->quantized_weightSize,
                      hipMemcpyHostToDevice,
                      stream);
      if (m->quantization_type == DT_INT4) {
        int parallelism = in_dim * out_dim / 2;
        decompress_int4_general_weights<DT>
            <<<GET_BLOCKS(parallelism),
               min(CUDA_NUM_THREADS, parallelism),
               0,
               stream>>>(m->quantized_weight_ptr,
                         static_cast<DT *>(m->weight_ptr),
                         in_dim,
                         in_dim * out_dim);
      } else {
        assert(m->quantization_type == DT_INT8);
        int parallelism = in_dim * out_dim;
        decompress_int8_general_weights<DT>
            <<<GET_BLOCKS(parallelism),
               min(CUDA_NUM_THREADS, parallelism),
               0,
               stream>>>(m->quantized_weight_ptr,
                         static_cast<DT *>(m->weight_ptr),
                         in_dim,
                         in_dim * out_dim);
      }

    } else {
      hipMemcpyAsync(m->weight_ptr,
                      weight_ptr,
                      in_dim * out_dim * sizeof(DT),
                      hipMemcpyHostToDevice,
                      stream);
    }
  }
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  DT alpha = 1.0f, beta = 0.0f;
  hipDataType input_type = ff_to_cuda_datatype(m->input_type[0]);
  hipDataType weight_type = m->offload
                                   ? ff_to_cuda_datatype(m->weight_ptr_type)
                                   : ff_to_cuda_datatype(m->weight_type[0]);
  hipDataType output_type = ff_to_cuda_datatype(m->output_type[0]);
  assert(input_type == weight_type && weight_type == output_type);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = input_type;
#endif
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
  cublasAlgoConfig_t mark{1, out_dim, batch_size, in_dim, (int)compute_type};
  auto iter = m->algo_map.find(mark);
  hipblasGemmAlgo_t algo_best =
      iter == m->algo_map.end() ? CUBLAS_GEMM_DEFAULT_TENSOR_OP
                                : static_cast<hipblasGemmAlgo_t>(iter->second);
#else
  hipblasGemmAlgo_t algo_best = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
#endif
  checkCUDA(hipblasGemmEx(m->handle.blas,
                         HIPBLAS_OP_T,
                         HIPBLAS_OP_N,
                         out_dim,
                         batch_size,
                         in_dim,
                         &alpha,
                         m->offload ? m->weight_ptr : weight_ptr,
                         weight_type,
                         in_dim,
                         input_ptr,
                         input_type,
                         in_dim,
                         &beta,
                         output_ptr,
                         output_type,
                         out_dim,
                         compute_type,
                         algo_best));
  // use_bias = True
  if (bias_ptr != NULL) {
#if defined(FF_USE_CUDA) || defined(FF_USE_HIP_CUDA)
    mark.k = 1;
    auto iter = m->algo_map.find(mark);
    algo_best = iter == m->algo_map.end()
                    ? CUBLAS_GEMM_DEFAULT_TENSOR_OP
                    : static_cast<hipblasGemmAlgo_t>(iter->second);
#else
    algo_best = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
#endif
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           out_dim,
                           batch_size,
                           1,
                           &alpha,
                           bias_ptr,
                           weight_type,
                           1,
                           static_cast<DT *>(m->one_ptr),
                           weight_type,
                           1,
                           &alpha,
                           output_ptr,
                           output_type,
                           out_dim,
                           compute_type,
                           algo_best));
  }
  if (use_activation(m->activation)) {
    checkCUDNN(hipdnnActivationForward(m->handle.dnn,
                                      m->actiDesc,
                                      &alpha,
                                      m->outputTensor,
                                      output_ptr,
                                      &beta,
                                      m->outputTensor,
                                      output_ptr));
  } else if (m->activation == AC_MODE_GELU) {
    size_t elements = (size_t)out_dim * (size_t)batch_size;
    constexpr float B = 0.7978845608028654f;   // sqrt(2.0/M_PI)
    constexpr float C = 0.035677408136300125f; // 0.044715 * sqrt(2.0/M_PI)
    gelu_forward_kernel<<<GET_BLOCKS(elements), CUDA_NUM_THREADS>>>(
        elements, B, C, (float *)output_ptr);
  } else if (m->activation == AC_MODE_NONE) {
    // Do nothing
  } else {
    assert(false && "Unsupported activation for Linear");
  }
}

template <typename DT>
void backward_kernel(LinearMeta const *m,
                     void const *input_ptr,
                     void *input_grad_ptr,
                     void const *output_ptr,
                     void *output_grad_ptr,
                     void const *kernel_ptr,
                     void *kernel_grad_ptr,
                     void *bias_grad_ptr,
                     int in_dim,
                     int out_dim,
                     int batch_size,
                     ffStream_t stream) {
  checkCUDA(hipblasSetStream(m->handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));

  DT alpha = 1.0f;
  float sgeam_alpha = 1.0f;
  hipDataType input_type = ff_to_cuda_datatype(m->input_type[0]);
  hipDataType weight_type = ff_to_cuda_datatype(m->weight_type[0]);
  hipDataType output_type = ff_to_cuda_datatype(m->output_type[0]);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  int output_size = out_dim * batch_size;
  if (m->activation == AC_MODE_RELU) {
    relu_backward_kernel(
        m->output_type[0], output_grad_ptr, output_ptr, output_size, stream);
  } else if (m->activation == AC_MODE_SIGMOID) {
    sigmoid_backward_kernel(
        m->output_type[0], output_grad_ptr, output_ptr, output_size, stream);
  } else {
    // TODO: only support relu and sigmoid for now
    assert(m->activation == AC_MODE_NONE);
  }
  // Compute weight gradiant
  // NOTE: we use alpha=1 for kernel_grad to accumulate gradients
  checkCUDA(hipblasGemmEx(m->handle.blas,
                         HIPBLAS_OP_N,
                         HIPBLAS_OP_T,
                         in_dim,
                         out_dim,
                         batch_size,
                         &alpha,
                         input_ptr,
                         input_type,
                         in_dim,
                         output_grad_ptr,
                         output_type,
                         out_dim,
                         &alpha,
                         kernel_grad_ptr,
                         weight_type,
                         in_dim,
                         compute_type,
                         CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  if (m->kernel_reg_type == REG_MODE_NONE) {
    // do nothing
  } else if (m->kernel_reg_type == REG_MODE_L2) {
    checkCUDA(hipblasSgeam(m->handle.blas,
                          HIPBLAS_OP_N,
                          HIPBLAS_OP_N,
                          in_dim,
                          out_dim,
                          &sgeam_alpha,
                          (float *)kernel_grad_ptr,
                          in_dim,
                          &(m->kernel_reg_lambda),
                          (float *)kernel_ptr,
                          in_dim,
                          (float *)kernel_grad_ptr,
                          in_dim));
  } else {
    assert(false && "Only L2 regularization is supported");
  }

  // Compute bias gradiant
  // NOTE: we use alpha=1 for bias_grad to accumulate gradients
  // use_bias = True
  if (bias_grad_ptr != NULL) {
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_T,
                           1,
                           out_dim,
                           batch_size,
                           &alpha,
                           static_cast<DT *>(m->one_ptr),
                           HIP_R_32F,
                           1,
                           output_grad_ptr,
                           output_type,
                           out_dim,
                           &alpha,
                           bias_grad_ptr,
                           weight_type,
                           1,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }
  // Compute data gradiant
  // NOTE: we use alpha=1 for input_grad to accumulate gradients
  if (input_grad_ptr != NULL) {
    checkCUDA(hipblasGemmEx(m->handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_N,
                           in_dim,
                           batch_size,
                           out_dim,
                           &alpha,
                           kernel_ptr,
                           weight_type,
                           in_dim,
                           output_grad_ptr,
                           output_type,
                           out_dim,
                           &alpha,
                           input_grad_ptr,
                           input_type,
                           in_dim,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }
}

template <typename DT>
__global__ void build_one_ptr(DT *one_ptr, int batch_size) {
  CUDA_KERNEL_LOOP(i, batch_size) {
    one_ptr[i] = static_cast<DT>(1.0f);
  }
}

} // namespace Internal
} // namespace Linear
} // namespace Kernels
} // namespace FlexFlow
